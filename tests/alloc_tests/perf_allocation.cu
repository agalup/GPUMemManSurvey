#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <algorithm>
#include <numeric>

#include "UtilityFunctions.cuh"
#include "PerformanceMeasure.cuh"
#include "DevicePerformanceMeasure.cuh"
#include "runtime_system.cuh"

#define DIRECT_MM_PTR

// ########################
#ifdef TEST_CUDA
#include "cuda/Instance.cuh"
using MemoryManager = MemoryManagerCUDA;
const std::string mem_name("CUDA");
#elif TEST_HALLOC
#include "halloc/Instance.cuh"
using MemoryManager = MemoryManagerHalloc;
const std::string mem_name("HALLOC");
#elif TEST_XMALLOC
#include "xmalloc/Instance.cuh"
using MemoryManager = MemoryManagerXMalloc;
const std::string mem_name("XMALLOC");
#elif TEST_SCATTERALLOC
#include "scatteralloc/Instance.cuh"
using MemoryManager = MemoryManagerScatterAlloc;
const std::string mem_name("ScatterAlloc");
#elif TEST_FDG
#include "fdg/Instance.cuh"
using MemoryManager = MemoryManagerFDG;
const std::string mem_name("FDGMalloc");
#elif TEST_OUROBOROS
#include "ouroboros/Instance.cuh"
	#ifdef TEST_PAGES
	#ifdef TEST_VIRTUALIZED_ARRAY
	using MemoryManager = MemoryManagerOuroboros<OuroVAPQ>;
	const std::string mem_name("Ouroboros-P-VA");
	#elif TEST_VIRTUALIZED_LIST
	using MemoryManager = MemoryManagerOuroboros<OuroVLPQ>;
	const std::string mem_name("Ouroboros-P-VL");
	#else
	using MemoryManager = MemoryManagerOuroboros<OuroPQ>;
	const std::string mem_name("Ouroboros-P-S");
	#endif
	#endif
	#ifdef TEST_CHUNKS
	#ifdef TEST_VIRTUALIZED_ARRAY
	using MemoryManager = MemoryManagerOuroboros<OuroVACQ>;
	const std::string mem_name("Ouroboros-C-VA");
	#elif TEST_VIRTUALIZED_LIST
	using MemoryManager = MemoryManagerOuroboros<OuroVLCQ>;
	const std::string mem_name("Ouroboros-C-VL");
	#else
	using MemoryManager = MemoryManagerOuroboros<OuroCQ>;
	const std::string mem_name("Ouroboros-C-S");
	#endif
	#endif
#elif TEST_REGEFF
#include "regeff/Instance.cuh"
	#ifdef TEST_ATOMIC
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::AtomicMalloc>;
	const std::string mem_name("RegEff-A");
	#elif TEST_ATOMIC_WRAP
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::AWMalloc>;
	const std::string mem_name("RegEff-AW");
	#elif TEST_CIRCULAR
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CMalloc>;
	const std::string mem_name("RegEff-C");
	#elif TEST_CIRCULAR_FUSED
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CFMalloc>;
	const std::string mem_name("RegEff-CF");
	#elif TEST_CIRCULAR_MULTI
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CMMalloc>;
	const std::string mem_name("RegEff-CM");
	#elif TEST_CIRCULAR_FUSED_MULTI
	using MemoryManager = MemoryManagerRegEff<RegEffVariants::CFMMalloc>;
	const std::string mem_name("RegEff-CFM");
	#endif
#endif

template <typename Runtime, bool warp_based>
__global__ void d_testAllocation_RS(Runtime rs, volatile int** verification_ptr, unsigned int num_allocations, unsigned int allocation_size)
{
	int tid{0};
	if(warp_based)
	{
		tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
		if(tid >= num_allocations)
			return;
		if(threadIdx.x % 32 == 0){
            rs.malloc((volatile int**)&verification_ptr[tid], allocation_size);
            assert(verification_ptr[tid]);
        }
	}
	else
	{
		tid = threadIdx.x + blockIdx.x * blockDim.x;
        if (tid == 0){
            printf("num allocations %d, allocation_size %d\n", num_allocations, allocation_size);
        }
		if(tid >= num_allocations)
			return;

        assert(verification_ptr);

        rs.malloc((volatile int**)&verification_ptr[tid], allocation_size);
        assert(verification_ptr[tid]);
        __threadfence();
        __syncthreads();
	}
}

template <typename Runtime>
__global__ void d_testAllocation_RS(Runtime rs, volatile int** verification_ptr, int num_allocations, int allocation_size, DevicePerfMeasure::Type* timing)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	DevicePerf perf_measure;
	
	// Start Measure
	perf_measure.startThreadMeasure();

    int* ptr;
    rs.malloc((volatile int**)&ptr, allocation_size);
	
	// Stop Measure
	timing[tid] = perf_measure.stopThreadMeasure();

	verification_ptr[tid] = ptr;
}

template <typename MemoryManagerType, bool warp_based>
__global__ void d_testFree_RS(Runtime<MemoryManagerType> rs, volatile int** verification_ptr, int num_allocations)
{
	int tid{0};
	if(warp_based)
	{
		tid = (threadIdx.x + blockIdx.x * blockDim.x) / 32;
		if(tid >= num_allocations)
			return;
	
		if(threadIdx.x % 32 == 0)
			rs.free(verification_ptr[tid]);
	}
	else
	{
		tid = threadIdx.x + blockIdx.x * blockDim.x;
		if(tid >= num_allocations)
			return;

		rs.free(verification_ptr[tid]);
	}
}

template <typename MemoryManagerType>
__global__ void d_testFree_RS(Runtime<MemoryManagerType> rs, volatile int** verification_ptr, int num_allocations, DevicePerfMeasure::Type* timing)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	DevicePerf perf_measure;

	// Start Measure
	perf_measure.startThreadMeasure();

	rs.free(verification_ptr[tid]);

	// Stop Measure
	timing[tid] = perf_measure.stopThreadMeasure();
}


__global__ void d_testWriteToMemory(volatile int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		ptr[i] = tid;
	}
}

__global__ void d_testReadFromMemory(volatile int** verification_ptr, int num_allocations, int allocation_size)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if(tid >= num_allocations)
		return;
	
	auto ptr = verification_ptr[tid];

	for(auto i = 0; i < (allocation_size / sizeof(int)); ++i)
	{
		if(ptr[i] != tid)
		{
			printf("%d | We got a wrong value here! %d vs %d\n", tid, ptr[i], tid);
			__trap();
		}
	}
}

int main(int argc, char* argv[])
{
	// Usage: <num_allocations> <size_of_allocation_in_byte> <num_iterations> <onDeviceMeasure> <warp-based> <generateoutput> <free_memory> <initial_path>
	unsigned int num_allocations{10000};
	unsigned int allocation_size_byte{8192};
	int num_iterations {100};
	bool warp_based{false};
	bool onDeviceMeasure{false};
	bool print_output{true};
	bool generate_output{false};
	bool free_memory{true};
	std::string alloc_csv_path{"../results/tmp/"};
	std::string free_csv_path{"../results/tmp/"};
	int allocSizeinGB{8};
	int device{0};
	if(argc >= 2)
	{
		num_allocations = atoi(argv[1]);
		if(argc >= 3)
		{
			allocation_size_byte = atoi(argv[2]);
			if(argc >= 4)
			{
				num_iterations = atoi(argv[3]);
				if(argc >= 5)
				{
					onDeviceMeasure = static_cast<bool>(atoi(argv[4]));
					if(argc >= 6)
					{
						warp_based = static_cast<bool>(atoi(argv[5]));
						if(onDeviceMeasure && warp_based)
						{
							std::cout << "OnDeviceMeasure and warp-based not possible at the same!" << std::endl;
							exit(-1);
						}
						if(argc >= 7)
						{
							generate_output = static_cast<bool>(atoi(argv[6]));
							if(argc >= 8)
							{
								free_memory = static_cast<bool>(atoi(argv[7]));
								if(argc >= 9)
								{
									alloc_csv_path = std::string(argv[8]);
									if(argc >= 10)
									{
										free_csv_path = std::string(argv[9]);
										if(argc >= 11)
										{
											allocSizeinGB = atoi(argv[10]);
											if(argc >= 12)
											{
												device = atoi(argv[11]);
											}
										}
									}
								}
							}
						}
					}
				}
			}
		}
    }

    debug("num_allocations %d\n", num_allocations);
    debug("allocation size %d\n", allocation_size_byte);
    debug("num_iterations %d\n", num_iterations);
    debug("onDeviceMeasure %d\n", onDeviceMeasure);
    debug("warp_based %d\n", warp_based);
    debug("generate_output %d\n", generate_output);
    debug("free memory %d\n", free_memory);

	allocation_size_byte = Utils::alignment(allocation_size_byte, sizeof(int));
	if(print_output)
		std::cout << "Number of Allocations: " << num_allocations << " | Allocation Size: " << allocation_size_byte << std::endl;

    /*int * tmp_dev;
    CHECK_ERROR(hipMalloc((void**)&tmp_dev, sizeof(int)));*/
	CHECK_ERROR(hipSetDevice(device));
	hipDeviceProp_t prop;
	CHECK_ERROR(hipGetDeviceProperties(&prop, device));
	std::cout << "Going to use " << prop.name << " " << prop.major << "." << prop.minor << "\n";

	std::cout << "--- " << mem_name << "---\n";

	volatile int** d_memory{nullptr};
	CHECK_ERROR(hipMalloc((void**)&d_memory, sizeof(volatile int*) * (num_allocations+100)));


	std::ofstream results_alloc, results_free;
	if(generate_output)
	{
		results_alloc.open(alloc_csv_path.c_str(), std::ios_base::app);
		results_free.open(free_csv_path.c_str(), std::ios_base::app);
	}

	int blockSize {256};
	int gridSize {Utils::divup<int>(num_allocations, blockSize)};
    printf("blockSize %d, gridSize %d\n", blockSize, gridSize);
	if(warp_based)
		gridSize *= 32;

	PerfMeasure timing_allocation;
	PerfMeasure timing_free;

	DevicePerfMeasure per_thread_timing_allocation(num_allocations, num_iterations);
	DevicePerfMeasure per_thread_timing_free(num_allocations, num_iterations);

    //exit(1);
	for(auto i = 0; i < num_iterations; ++i)
    {
        debug("iteration %d\n", i);

#ifdef DIRECT_MM_PTR
    #ifdef TEST_OUROBOROS
        debug("mm with direct mm ptr type\n");
	    MemoryManager memory_manager(allocSizeinGB * 1024ULL * 1024ULL * 1024ULL);
        using MemoryManager2 = std::remove_pointer<decltype(memory_manager.d_memory_manager)>::type;
    #else
        debug("cuda mm\n");
	    MemoryManager* memory_manager;
        GUARD_CU(hipMallocManaged((void**)&memory_manager, sizeof(MemoryManager)));
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        new (memory_manager) MemoryManager(allocSizeinGB * 1024ULL * 1024ULL * 1024ULL);
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        using MemoryManager2 = MemoryManager;
    #endif
#else
        debug("mm without direct mm ptr type\n");
	    MemoryManager* memory_manager;
        GUARD_CU(hipMallocManaged((void**)&memory_manager, sizeof(MemoryManager)));
        debug("mm, memory allocated, init to be\n");
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        new (memory_manager) MemoryManager(allocSizeinGB * 1024ULL * 1024ULL * 1024ULL);
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        debug("[%s:%d]mm address %x \n", __FUNCTION__, __LINE__, memory_manager);
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        //debug("[%s:%d]malloc address %x \n", __FUNCTION__, __LINE__, memory_manager->malloc);
        fflush(stdout);
        
	    //MemoryManager memory_manager(allocSizeinGB * 1024ULL * 1024ULL * 1024ULL);
        using MemoryManager2 = MemoryManager;
#endif
        Runtime<MemoryManager2> rs;
        int app_sm = 70;

#ifdef DIRECT_MM_PTR
    #ifdef TEST_OUROBOROS
        #ifdef CALLBACK__
            debug("RS with direct ptr to mm and callback!\n");
            rs.init(num_allocations, 0, memory_manager.d_memory_manager, 1, app_sm, 5, 4, 1, blockSize, 1);
        #else
            debug("RS with direct ptr to mm\n");
            rs.init(num_allocations, 0, memory_manager.d_memory_manager, app_sm, 5, 4, blockSize, 1);
        #endif
    #else
        #ifdef CALLBACK__
            debug("RS with direct ptr to mm and callback!\n");
            rs.init(num_allocations, 0, memory_manager, app_sm, 5, 4, 1, blockSize, 1);
        #else
            debug("RS with direct ptr to mm\n");
            rs.init(num_allocations, 0, memory_manager, app_sm, 5, 4, blockSize, 1);
        #endif
    #endif
#else
    #ifdef CALLBACK__
        debug("RS with callback\n");
        rs.init(num_allocations, 0, memory_manager, 1, app_sm, 5, 4, 1, blockSize, 1);
    #else
        debug("RS, no direct ptr to mm and no callback\n");
        rs.init(num_allocations, 0, memory_manager, app_sm, 5, 4, blockSize, 1);
    #endif
#endif
        hipCtx_t app_ctx; 
        CUexecAffinityParam_v1 app_param{CUexecAffinityType::CU_EXEC_AFFINITY_TYPE_SM_COUNT, (unsigned int) app_sm};
        auto affinity_flags = CUctx_flags::hipDeviceScheduleAuto;
        GUARD_CU((hipError_t)cuCtxCreate_v3(&app_ctx, &app_param, 1, affinity_flags, device));
        GUARD_CU((hipError_t)hipCtxSynchronize());
        hipCtx_t current_ctx;
        GUARD_CU((hipError_t)hipCtxPopCurrent(&current_ctx));
        debug("current was %d\n", current_ctx);

        std::cout << "#" << std::flush;

        GUARD_CU((hipError_t)hipCtxSynchronize());

        if(onDeviceMeasure)
        {
            void* args[] = {&rs, &d_memory, &num_allocations, &allocation_size_byte, per_thread_timing_allocation.getDevicePtr()};
            rs.run_sync((void*)d_testAllocation_RS<Runtime<MemoryManager2>>, gridSize, blockSize, args, app_ctx);
            CHECK_ERROR(hipDeviceSynchronize());
            per_thread_timing_allocation.acceptResultsFromDevice();
        }
        else
        {
            void* args[] = {&rs, &d_memory, &num_allocations, &allocation_size_byte};
            timing_allocation.startMeasurement();
            if(warp_based){
                rs.run_sync((void*)d_testAllocation_RS<Runtime<MemoryManager2>, true>, gridSize, blockSize, args, app_ctx);
            }else{
                rs.run_sync((void*)d_testAllocation_RS<Runtime<MemoryManager2>, false>, gridSize, blockSize, args, app_ctx);
            }
            timing_allocation.stopMeasurement();
            CHECK_ERROR(hipDeviceSynchronize());
        }
        debug("write\n");
        void* args2[] = {&d_memory, &num_allocations, &allocation_size_byte};
        rs.run_sync((void*)d_testWriteToMemory, gridSize, blockSize, args2, app_ctx);
        debug("read\n");
        void* args3[] = {&d_memory, &num_allocations, &allocation_size_byte};
        rs.run_sync((void*)d_testReadFromMemory, gridSize, blockSize, args3, app_ctx);
        debug("free\n");
        if(free_memory)
        {
            if(onDeviceMeasure)
            {
                void* args[] = {&rs, &d_memory, &num_allocations, per_thread_timing_allocation.getDevicePtr()};
                rs.run_sync((void*)d_testFree_RS<Runtime<MemoryManager2>>, gridSize, blockSize, args, app_ctx);
                //d_testFree_RS <<<gridSize, blockSize>>>(rs, d_memory, num_allocations, per_thread_timing_free.getDevicePtr());
                CHECK_ERROR(hipDeviceSynchronize());
                per_thread_timing_free.acceptResultsFromDevice();
            }
            else
            {
                void* args[] = {&rs, &d_memory, &num_allocations};
                timing_free.startMeasurement();
                if(warp_based){
                    rs.run_sync((void*)d_testFree_RS<Runtime<MemoryManager2>, true>, gridSize, blockSize, args, app_ctx);
                    //d_testFree_RS <MemoryManager2, true> <<<gridSize, blockSize>>>(rs, d_memory, num_allocations);
                }else{
                    rs.run_sync((void*)d_testFree_RS<Runtime<MemoryManager2>, false>, gridSize, blockSize, args, app_ctx);
                }
                timing_free.stopMeasurement();
                CHECK_ERROR(hipDeviceSynchronize());
            }
        }

        debug("stop services\n");
        rs.stop_services();
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        debug("clean memory\n");
        clean_memory(app_sm, blockSize, rs);
        GUARD_CU((hipError_t)hipCtxDestroy(app_ctx));
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        debug("memory cleaned\n");
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());
        debug("stop runtime\n");
        rs.stop_runtime();
        GUARD_CU(hipDeviceSynchronize());
        GUARD_CU(hipPeekAtLastError());

	}
	std::cout << std::endl;

	if(onDeviceMeasure)
	{
		auto alloc_result = per_thread_timing_allocation.generateResult();
		auto free_result = per_thread_timing_free.generateResult();

		if(print_output)
		{
			std::cout << "Timing Allocation: Mean:" << alloc_result.mean_ << "cycles | Median: " << alloc_result.median_ << " ms" << std::endl;
			std::cout << "Timing       Free: Mean:" << free_result.mean_ << "cycles | Median: " << free_result.median_ << " ms" << std::endl;
		}
		if(generate_output)
		{
			results_alloc << alloc_result.mean_ << "," << alloc_result.std_dev_ << "," << alloc_result.min_ << "," << alloc_result.max_ << "," << alloc_result.median_;
			results_free << free_result.mean_ << "," << free_result.std_dev_ << "," << free_result.min_ << "," << free_result.max_ << "," << free_result.median_;
		}
	}
	else
	{
		auto alloc_result = timing_allocation.generateResult();
		auto free_result = timing_free.generateResult();
		if(print_output)
		{
			std::cout << "Timing Allocation: Mean:" << alloc_result.mean_ << "ms" << std::endl;// " | Median: " << alloc_result.median_ << " ms" << std::endl;
			std::cout << "Timing       Free: Mean:" << free_result.mean_ << "ms" << std::endl;// "  | Median: " << free_result.median_ << " ms" << std::endl;
		}
		if(generate_output)
		{
			results_alloc << alloc_result.mean_ << "," << alloc_result.std_dev_ << "," << alloc_result.min_ << "," << alloc_result.max_ << "," << alloc_result.median_;
			results_free << free_result.mean_ << "," << free_result.std_dev_ << "," << free_result.min_ << "," << free_result.max_ << "," << free_result.median_;
		}
	}
	
	return 0;
}
